#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

using uint8_t = unsigned char;
using uint32_t = unsigned int;

#include "../state_word.h"
#include "../trajectory_status.h"

__device__ int select_flip_bit(int state_size, const float* __restrict__ transition_rates, float total_rate,
							   hiprandState* __restrict__ rand)
{
	float r = hiprand_uniform(rand) * total_rate;
	float sum = 0;
	int idx = 0;
	for (int i = 0; i < state_size; i++)
	{
		sum += transition_rates[i];
		idx += (sum < r) ? 1 : 0;
	}
	return idx;
}

extern "C" __global__ void initialize_random(int trajectories_count, unsigned long long seed,
											 hiprandState* __restrict__ rands)
{
	auto id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= trajectories_count)
		return;

	// initialize random number generator
	hiprand_init(seed, id, 0, rands + id);
}

extern "C" __global__ void initialize_initial_state(int trajectories_count, int state_size,
													const float* __restrict__ initial_probs,
													state_word_t* __restrict__ states, float* __restrict__ times,
													hiprandState* __restrict__ rands)
{
	auto id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= trajectories_count)
		return;

	constexpr int word_size = sizeof(state_word_t) * 8;
	int state_words = (state_size + word_size - 1) / word_size;

	// initialize state
	{
		state_word_t s = 0;

		// randomly set free vars
		for (int i = 0; i < state_size; i++)
		{
			if (hiprand_uniform(rands + id) <= initial_probs[i])
				s |= 1 << (i % word_size);
			else
				s &= ~(1 << (i % word_size));

			if (i % word_size == word_size - 1)
			{
				states[id * state_words + i / word_size] = s;
				s = 0;
			}
		}

		if (state_size % word_size != 0)
			states[id * state_words + state_words - 1] = s;
	}

	// set time to zero
	times[id] = 0.f;
}

extern __device__ float compute_transition_rates(float* __restrict__ transition_rates,
												 const state_word_t* __restrict__ state);
extern __device__ float compute_transition_entropy(const float* __restrict__ transition_rates);

__device__ void simulate_inner(int trajectories_count, int state_size, int trajectory_limit, float time_tick,
							   float max_time, bool discrete_time, state_word_t* __restrict__ last_states,
							   float* __restrict__ last_times, void* __restrict__ rands_v,
							   state_word_t* __restrict__ trajectory_states, float* __restrict__ trajectory_times,
							   float* __restrict__ trajectory_transition_entropies,
							   trajectory_status* __restrict__ trajectory_statuses,
							   float* __restrict__ transition_rates, state_word_t* __restrict__ state)
{
	hiprandState* __restrict__ rands = reinterpret_cast<hiprandState*>(rands_v);
	auto id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= trajectories_count)
		return;

	constexpr int word_size = sizeof(state_word_t) * 8;
	int state_words = (state_size + word_size - 1) / word_size;

	for (int i = 0; i < state_words; i++)
		state[i] = last_states[id * state_words + i];
	hiprandState rand = rands[id];
	float time = last_times[id];
	int step = 0;
	trajectory_states = trajectory_states + id * state_words * trajectory_limit;
	trajectory_times = trajectory_times + id * trajectory_limit;
	trajectory_transition_entropies = trajectory_transition_entropies + id * trajectory_limit;
	trajectory_status status = trajectory_status::CONTINUE;

	// as the first time set the last from the prev run
	trajectory_times[step++] = time;

	while (true)
	{
		// get transition rates for current state
		float total_rate = compute_transition_rates(transition_rates, state);

		float transition_entropy = 0.f;

		// if total rate is zero, no transition is possible
		if (total_rate == 0.f)
		{
			status = trajectory_status::FIXED_POINT;
			time = max_time;
		}
		else
		{
			if (discrete_time)
			{
				time += time_tick;
				// due to a common rounding error for discrete simulations,
				// we set time to max_time explicitely if it is close enough to max_time
				time = max_time - time_tick < time ? max_time : time;
			}
			else
				time += -logf(hiprand_uniform(&rand)) / total_rate;

			time = fminf(time, max_time);

			// if total rate is nonzero, we compute the transition entropy
			transition_entropy = compute_transition_entropy(transition_rates);
		}

		for (int i = 0; i < state_words; i++)
			trajectory_states[step * state_words + i] = state[i];
		trajectory_times[step] = time;
		trajectory_transition_entropies[step] = transition_entropy;
		step++;

		if (time >= max_time || step >= trajectory_limit)
			break;

		int flip_bit = select_flip_bit(state_size, transition_rates, total_rate, &rand);
		state[flip_bit / word_size] ^= 1 << (flip_bit % word_size);
	}

	// save thread variables
	{
		for (int i = 0; i < state_words; i++)
			last_states[id * state_words + i] = state[i];
		rands[id] = rand;
		last_times[id] = time;
	}

	if (status != trajectory_status::FIXED_POINT)
	{
		status = (time >= max_time) ? trajectory_status::FINISHED : trajectory_status::CONTINUE;
	}

	trajectory_statuses[id] = status;
}
